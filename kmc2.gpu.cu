#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <sstream>
#include <algorithm>
#include <list>
#include <vector>
#include <sys/time.h>
#include <stdint.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#include "mmer.h"
#include "kmer.h"
#include "kxmer_set.h"

#define DEVE
#define STATS_FASTQ_SIZE (1 << 27)
#define STATS_READ_NUM 1278085
#define READS_PER_LOAD (1 << 21)
#define READ_LENGTH 100
#define NUM_BYTES_PER_READ 32
#define NUM_BLOCKS 2497
#define NUM_THREADS_PER_BLOCK 128
#define SIGNATURE_LEN 7
#define KMER_LENGTH 50
#define MAP_SIZE ((1 << 2 * SIGNATURE_LEN) + 1)
#define CODES_SIZE 256
#define BIN_PART_SIZE (1 << 16)
#define NUM_PARTS 38
#define BIN_NO 512
#define TEMP_FILE_BUFFER (24 * (1 << 20))
#define POS_ARRAY_SIZE 1024
#define MAX_K 256
#define KMER_WORDS 2
#define KMER_X 3
#define ALIGNMENT 0x100
#define EXPAND_BUFFER_RECS (1 << 16)
#define CUTOFF_MIN 2
#define CUTOFF_MAX 1000000000
#define NUM_STREAM 4	

#define MIN(x,y) ((x) < (y) ? (x) : (y))
#define MAX(x,y) ((x) > (y) ? (x) : (y))
#define NORM(x, lower, upper)	((x) < (lower) ? (lower) : (x) > (upper) ? (upper) : (x))
#define BYTE_LOG(x) (((x) < (1 << 8)) ? 1 : ((x) < (1 << 16)) ? 2 : ((x) < (1 << 24)) ? 3 : 4)
#define USE_NVTX

#ifdef USE_NVTX
#include "nvToolsExt.h"
const uint32_t colors[] = { 0x0000ff00, 0x000000ff, 0x00ffff00, 0x00ff00ff, 0x0000ffff, 0x00ff0000, 0x00ffffff };
const int num_colors = sizeof(colors)/sizeof(uint32_t);

#define PUSH_RANGE(name,cid) { \
    int color_id = cid; \
    color_id = color_id%num_colors;\
    nvtxEventAttributes_t eventAttrib = {0}; \
    eventAttrib.version = NVTX_VERSION; \
    eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
    eventAttrib.colorType = NVTX_COLOR_ARGB; \
    eventAttrib.color = colors[color_id]; \
    eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
    eventAttrib.message.ascii = name; \
    nvtxRangePushEx(&eventAttrib); \
}
#define POP_RANGE nvtxRangePop();
#else
#define PUSH_RANGE(name,cid)
#define POP_RANGE
#endif

using namespace std;

__constant__ char code[CODES_SIZE];

struct Cmp {
__host__ __device__  bool operator()(const CKmer &x, const CKmer &y) {
	for(int i = 1; i >= 0; --i)
		if(x.data[i] < y.data[i])
			return true;
		else if(x.data[i] > y.data[i])
			return false;
	return false;
	}
};


class binDesc
{
	public:
	int64_t bin_desc_size[BIN_NO];
	uint64_t bin_desc_n_rec[BIN_NO];
	uint64_t bin_desc_n_plus_x_recs[BIN_NO];
	uint64_t bin_desc_n_super_kmers[BIN_NO];
};

class completerDesc
{
	public: 
	unsigned char* data[BIN_NO];
	uint64_t data_size[BIN_NO];
	unsigned char* lut[BIN_NO];
	uint64_t lut_size[BIN_NO];
	uint64_t _n_unique[BIN_NO];
	uint64_t _n_cutoff_min[BIN_NO];
	uint64_t _n_cutoff_max[BIN_NO];
	uint64_t _n_total[BIN_NO];
};

class Comp
{
	uint32_t* signature_occurences;
	public:
		__host__ Comp(uint32_t* _signature_occurences) : signature_occurences(_signature_occurences){}
		__host__ bool operator()(int i, int j)
		{
			return signature_occurences[i] > signature_occurences[j];
		}
};


__host__ void GetNextSymb(unsigned char& symb, unsigned char& byte_shift, uint64_t& pos, unsigned char* data_p)
{
	symb = (data_p[pos] >> byte_shift) & 3;
//	printf("line 223 reached\n");
	if (byte_shift == 0)
	{
		pos++;
		byte_shift = 6;
	}
	else
		byte_shift -= 2;
//	printf("pos is: %d\n", pos);
}


__host__ void FromChildThread(CKmer *input_array , CKmer* thread_buffer, uint64_t size, uint32_t& input_pos)
{
	memcpy(input_array + input_pos, thread_buffer, size * (sizeof(CKmer)));
	//for(int i = 0; i < size; i++)
	//	H.push_back(thread_buffer[i]);
	input_pos += (uint32_t)size;
}

__host__ void expandKxmers(unsigned char *file_buffer, CKmer *input_array, uint64_t start_pos, uint64_t end_pos, uint32_t max_x, unsigned char* lut, uint32_t& input_pos) 
{
	//	printf("function being called\n");
		unsigned char _raw_buffer[1 << 21];
		CKmer* buffer = (CKmer*)_raw_buffer;

		CKmer kmer, rev_kmer, kmer_mask;
		CKmer  kxmer_mask;
		bool kmer_lower; //true if kmer is lower than its rev. comp
		uint32_t x, additional_symbols;
		unsigned char symb;
		uint32_t kmer_bytes = (KMER_LENGTH + 3) / 4;
		uint32_t rev_shift = KMER_LENGTH * 2 - 2;
		unsigned char *data_p = file_buffer;
		kmer_mask.set_n_1(KMER_LENGTH * 2);
		uint32_t kmer_shr = KMER_WORDS * 32 - KMER_LENGTH;

		kxmer_mask.set_n_1((KMER_LENGTH + max_x + 1) * 2);

		uint64_t buffer_pos = 0;
		uint64_t pos = start_pos;	

	//	printf("start is: %u, end is: %u \n", start_pos, end_pos);

		while (pos < end_pos)
		{
			kmer.clear();
			rev_kmer.clear();

			additional_symbols = data_p[pos++];

			//building kmer
			for (uint32_t i = 0, kmer_pos = 8 * KMER_WORDS - 1, kmer_rev_pos = 0; (int)i < (int)kmer_bytes; ++i, --kmer_pos, ++kmer_rev_pos)
			{
				kmer.set_byte(kmer_pos, data_p[pos + i]);
				rev_kmer.set_byte(kmer_rev_pos, lut[data_p[pos + i]]);
			}
			pos += (uint64_t)kmer_bytes;
			unsigned char byte_shift = 6 - (KMER_LENGTH % 4) * 2;
			if (byte_shift != 6)
				--pos;

			if (kmer_shr)
				kmer.SHR(kmer_shr);

			kmer.mask(kmer_mask);
			rev_kmer.mask(kmer_mask);

			kmer_lower = kmer < rev_kmer;
			x = 0;
			if (kmer_lower)
				buffer[buffer_pos].set(kmer);
			else
				buffer[buffer_pos].set(rev_kmer);

		//	printf("buffer_pos is: %d; pos is: %d; end_pos is: %d \n", buffer_pos, pos, end_pos);


			uint32_t symbols_left = additional_symbols;
			
			//printf("symbols_left is: %u\n", symbols_left);
			while (symbols_left != 0)
			{
		//		printf("line 299 reached\n");
				GetNextSymb(symb, byte_shift, pos, data_p);
				kmer.SHL_insert_2bits(symb);
				kmer.mask(kmer_mask);
				rev_kmer.SHR_insert_2bits(3 - symb, rev_shift);
				--symbols_left;

		//		printf("line 303 reached\n");

		//		printf("309 pos is: %ld\n", pos);
				if (kmer_lower)
				{
					if (kmer < rev_kmer)
					{
						buffer[buffer_pos].SHL_insert_2bits(symb);
						++x;
						if (x == max_x)
						{
							if (symbols_left == 0)
								break;

							buffer[buffer_pos++].set_2bits(x, KMER_LENGTH * 2 + max_x * 2);
							if((int)buffer_pos >= EXPAND_BUFFER_RECS)
							{
								FromChildThread(input_array, buffer, buffer_pos, input_pos);
								buffer_pos = 0;
							}
							x = 0;

							GetNextSymb(symb, byte_shift, pos, data_p);
							kmer.SHL_insert_2bits(symb);
							kmer.mask(kmer_mask);
							rev_kmer.SHR_insert_2bits(3 - symb, rev_shift);
							--symbols_left;

							kmer_lower = kmer < rev_kmer;

							if (kmer_lower)
								buffer[buffer_pos].set(kmer);
							else
								buffer[buffer_pos].set(rev_kmer);
						}
		//				printf("line 338 reached\n");
					}
					else
					{
						buffer[buffer_pos++].set_2bits(x, KMER_LENGTH * 2 + max_x * 2);
						if ((int)buffer_pos >= EXPAND_BUFFER_RECS)
						{
							FromChildThread(input_array, buffer, buffer_pos, input_pos);
							buffer_pos = 0;
						}
						x = 0;

						kmer_lower = false;
						buffer[buffer_pos].set(rev_kmer);
		//				printf("line 352 reached\n");
					}
				}
				else
				{
					if (!(kmer < rev_kmer))
					{
						buffer[buffer_pos].set_2bits(3 - symb, KMER_LENGTH * 2 + x * 2);
						++x;
						if (x == max_x)
						{
							if (symbols_left == 0)
								break;

							buffer[buffer_pos++].set_2bits(x, KMER_LENGTH * 2 + max_x * 2);
							if ((int)buffer_pos >= EXPAND_BUFFER_RECS)
							{
								FromChildThread(input_array, buffer, buffer_pos, input_pos);
								buffer_pos = 0;
							}
							x = 0;

							GetNextSymb(symb, byte_shift, pos, data_p);
							kmer.SHL_insert_2bits(symb);
							kmer.mask(kmer_mask);
							rev_kmer.SHR_insert_2bits(3 - symb, rev_shift);
							--symbols_left;

							kmer_lower = kmer < rev_kmer;

							if (kmer_lower)
								buffer[buffer_pos].set(kmer);
							else
								buffer[buffer_pos].set(rev_kmer);
						}
		//				printf("line 387 reached\n");
					}
					else
					{
						buffer[buffer_pos++].set_2bits(x, KMER_LENGTH * 2 + max_x * 2);
						if ((int)buffer_pos >= EXPAND_BUFFER_RECS)
						{
							FromChildThread(input_array, buffer, buffer_pos, input_pos);
							buffer_pos = 0;
						}
						x = 0;

						buffer[buffer_pos].set(kmer);
						kmer_lower = true;
					}
		//			printf("line 402 reached\n");
				}
		//		printf("407 pos is: %ld\n", pos);
				
			}
		//	printf("line 406 reached\n");
			buffer[buffer_pos++].set_2bits(x, KMER_LENGTH * 2 + max_x * 2);
			if ((int)buffer_pos >= EXPAND_BUFFER_RECS)
			{
				FromChildThread(input_array, buffer, buffer_pos, input_pos);
				buffer_pos = 0;
			}
			if (byte_shift != 6)
				++pos;
		//	printf("419 pos is: %ld\n", pos);
		//	printf("line 414 reached\n");
		}
		if (buffer_pos > 0)
		{
			FromChildThread(input_array, buffer, buffer_pos, input_pos);
			buffer_pos = 0;
		}


	}
	
	__host__ void initfiles(FILE** files) {
		string f_name;
		
		for(int i = 0; i < BIN_NO; i++) {
			string s_tmp;
			stringstream convert;
			convert << i;
			s_tmp = convert.str();
			while(s_tmp.length() < 5) {
				s_tmp = string("0") + s_tmp;
			}
			f_name = "kmc_" + s_tmp + ".bin";
			files[i] = fopen(f_name.c_str(), "wb+");
			fclose(files[i]);
		}

	}

	__host__ void openfiles(FILE** files)
	{
		string f_name;
		
		for(int i = 0; i < BIN_NO; i++) {
			string s_tmp;
			stringstream convert;
			convert << i;
			s_tmp = convert.str();
			while(s_tmp.length() < 5) {
				s_tmp = string("0") + s_tmp;
			}
			f_name = "kmc_" + s_tmp + ".bin";
			files[i] = fopen(f_name.c_str(), "ab+");
		}
	};

	__host__ void deletefiles()
        {
                string f_name;

                for(int i = 0; i < BIN_NO; i++) {
                        string s_tmp;
                        stringstream convert;
                        convert << i;
                        s_tmp = convert.str();
                        while(s_tmp.length() < 5) {
                                s_tmp = string("0") + s_tmp;
                        }
                        f_name = "kmc_" + s_tmp + ".bin";
                        remove(f_name.c_str());
                }
        };


	__host__ void closefiles(FILE** files) 
	{
		for(int i = 0; i < BIN_NO; i++) {
			fclose(files[i]);
		}	
	}

	__host__ void rewindfiles(FILE** files) 
	{
		string f_name;
		
		for(int i = 0; i < BIN_NO; i++) {
			string s_tmp;
			stringstream convert;
			convert << i;
			s_tmp = convert.str();
			while(s_tmp.length() < 5) {
				s_tmp = string("0") + s_tmp;
			}
			f_name = "kmc_" + s_tmp + ".bin";
			files[i] = fopen(f_name.c_str(), "rb+");
		}

	}
	
	__host__ void init_bin_desc(binDesc &bin_desc)
	{
		for(int i = 0; i < BIN_NO; i++) {
			bin_desc.bin_desc_size[i] = 0;
			bin_desc.bin_desc_n_rec[i] = 0;
			bin_desc.bin_desc_n_plus_x_recs[i] = 0;
			bin_desc.bin_desc_n_super_kmers[i] = 0;
		}	
	}
	
	__host__ void putBinsToDisk(FILE** files, uint64_t &tmp_size, binDesc &bin_desc, unsigned char* buffer, int64_t* bin_desc_size
				    , uint64_t* bin_desc_n_rec, uint32_t* bin_desc_n_plus_x_recs, uint64_t* bin_desc_n_super_kmers, int buffer_size, int bin_size)
	{
		PUSH_RANGE("fwrite", 1)
		openfiles(files);
		//#pragma omp parallel for
		for(int stream = 0; stream < NUM_STREAM; stream++) {
			unsigned char* stream_buffer = buffer + stream * buffer_size;
			int offset = stream * BIN_NO;
			for(int i = 0; i < BIN_NO; i++) {
				if(bin_desc_size[i + offset] != 0)
					fwrite(stream_buffer + i * bin_size, 1, bin_desc_size[i + offset], files[i]);	
				tmp_size += bin_desc_size[i + offset];
				bin_desc.bin_desc_size[i] += bin_desc_size[i + offset];
				bin_desc.bin_desc_n_rec[i] += bin_desc_n_rec[i + offset];
				bin_desc.bin_desc_n_plus_x_recs[i] += bin_desc_n_plus_x_recs[i + offset];
				bin_desc.bin_desc_n_super_kmers[i] += bin_desc_n_super_kmers[i + offset];
			}
		}
		closefiles(files);
		POP_RANGE
	}
	
	__host__ void calc_codes(char *codes) {
		for(int i = 0; i < 256; i++)
			codes[i] = -1;
		codes['A'] = codes['a'] = 0;
		codes['C'] = codes['c'] = 1;
		codes['G'] = codes['g'] = 2;
		codes['T'] = codes['t'] = 3;
	}

	__host__ uint64_t round_up_to_alignment(int64_t x)
	{
		return (x + ALIGNMENT-1) / ALIGNMENT * ALIGNMENT;
	}




	__host__ void init(uint32_t* stats, int * signature_map) {

		int n_bins = BIN_NO;
		uint32_t sorted[MAP_SIZE];
		for(uint32_t i = 0; i < MAP_SIZE; i++)
		{
			sorted[i] = i;
		}
		sort(sorted, sorted + MAP_SIZE, Comp(stats));

	/*	FILE* out;
		out = fopen("sorted.txt","w");
		
		for(int i = 0; i < (1 << SIGNATURE_LEN * 2) + 1; i++) {
			fprintf(out, "%d: %d\n", i, sorted[i]);
		}
		fclose(out);	
	*/

		list<uint32_t> _stats0;
		list<uint64_t> _stats1;	

		for (uint32_t i = 0; i < MAP_SIZE ; ++i)
		{
			if (CMmer::is_allowed(sorted[i], SIGNATURE_LEN))
			{
				_stats0.push_back(sorted[i]);
				_stats1.push_back(stats[sorted[i]]);
			}
		}

		list<uint32_t> group0;
		list<uint64_t> group1;
	//	uint32_t groupStartPos = 0;
	//	uint32_t groupEndPos = 0;	

		int bin_no = 0;
		//counting sum
		double sum = 0.0;
		for (list<uint64_t>::iterator i = _stats1.begin(); i != _stats1.end(); i++)
		{
			*i += (uint64_t)1000;
			sum += (double)*i;
		}

		double mean = sum / n_bins;
		double max_bin_size = 1.1 * mean;
		uint32_t n = n_bins - 1; //one is needed for disabled signatures
		uint32_t max_bins = n_bins - 1;

		while (_stats1.size() > n)
		{
			uint32_t max0 = _stats0.front();
			uint64_t max1 = _stats1.front();

			if (max1 > mean)
			{
				signature_map[max0] = bin_no++;				
				sum -= max1;
				mean = sum / (max_bins - bin_no);
				max_bin_size = 1.1 * mean;

				_stats0.pop_front();
				_stats1.pop_front();
				--n;
			}
			else
			{
				//heuristic
				group0.clear();
				group1.clear();
				double tmp_sum = 0.0;
				uint32_t in_current = 0;
				list<uint64_t>::iterator it1 = _stats1.begin();
				for (list<uint32_t>::iterator it0 = _stats0.begin(); it0 != _stats0.end();)
				{
					if (tmp_sum + *it1 < max_bin_size)
					{
						tmp_sum += *it1;
						group0.push_back(*it0);
						group1.push_back(*it1);
						it0 = _stats0.erase(it0);
						it1 = _stats1.erase(it1);
						++in_current;
					}
					else
					{	
						++it0;
						++it1;
					}
				}

				list<uint32_t>::iterator i0 = group0.begin();
				list<uint64_t>::iterator i1 = group1.begin();
				for (; i0 != group0.end(); ++i0)
				{
					signature_map[*i0] = bin_no;
				}
				--n;
				++bin_no;

				sum -= tmp_sum;
				mean = sum / (max_bins - bin_no);
				max_bin_size = 1.1 * mean;
			}
		}

		if (_stats0.size() > 0)
		{
			for (list<uint32_t>::iterator i = _stats0.begin(); i != _stats0.end(); ++i)
			{
				signature_map[*i] = bin_no++;
				//cout << "rest bin: " << i->second << "\n";
			}
		}
		signature_map[1 << 2 * SIGNATURE_LEN] = bin_no;



	}


	__global__ void calcStats(char* readList, uint32_t* statsBin, uint32_t* norm7)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if(tid >= STATS_READ_NUM)
			return;
		char* read = readList + tid * NUM_BYTES_PER_READ * 4;
		
		char seq[READ_LENGTH];
		uint32_t signatureStartPos;

		CMmer currentSignature, endMmer;

		char c;
		for(int i = 0; i < READ_LENGTH; i++) {
			c = read[i];
			if(c < 32)
				break;
			seq[i] = code[c];
		}	

		uint32_t i;
		uint32_t len;//length of extended kmer
		

		i = 0;
		len = 0;
		while (i + KMER_LENGTH - 1 < READ_LENGTH)
		{	
			bool contains_N = false;
			//building first signature after 'N' or at the read begining
			for (uint32_t j = 0; j < SIGNATURE_LEN; ++j, ++i)
			if (seq[i] < 0)//'N'
			{
				contains_N = true;
				break;
			}
			//signature must be shorter than k-mer so if signature contains 'N', k-mer will contains it also
			if (contains_N)
			{
				++i;
				continue;
			}
			len = SIGNATURE_LEN;
			signatureStartPos = i - SIGNATURE_LEN;
			currentSignature.insert(seq + signatureStartPos, norm7);
			endMmer.set(currentSignature);
			for (; i < READ_LENGTH; ++i)
			{
				if (seq[i] < 0)//'N'
				{
					if (len >= KMER_LENGTH)
						atomicAdd(&(statsBin[currentSignature.get()]), 1 + len - KMER_LENGTH);
					len = 0;
					++i;
					break;
				}
				endMmer.insert(seq[i], norm7);
				if (endMmer < currentSignature)//signature at the end of current k-mer is lower than current
				{
					if (len >= KMER_LENGTH)
					{
						atomicAdd(&(statsBin[currentSignature.get()]), 1 + len - KMER_LENGTH);
						len = KMER_LENGTH - 1;
					}
					currentSignature.set(endMmer);
					signatureStartPos = i - SIGNATURE_LEN + 1;
				}
				else if (endMmer == currentSignature)
				{
					currentSignature.set(endMmer);
					signatureStartPos = i - SIGNATURE_LEN + 1;
				}
				else if (signatureStartPos + KMER_LENGTH - 1 < i)//need to find new signature
				{
					atomicAdd(&(statsBin[currentSignature.get()]), 1 + len - KMER_LENGTH);
					len = KMER_LENGTH - 1;
					//looking for new signature
					++signatureStartPos;
					//building first signature in current k-mer
					endMmer.insert(seq + signatureStartPos, norm7);
					currentSignature.set(endMmer);
					for (uint32_t j = signatureStartPos + SIGNATURE_LEN; j <= i; ++j)
					{
						endMmer.insert(seq[j], norm7);
						if (endMmer <= currentSignature)
						{
							currentSignature.set(endMmer);
							signatureStartPos = j - SIGNATURE_LEN + 1;
						}
					}
				}
				++len;
			}
		}
		if (len >= KMER_LENGTH)//last one in read
			atomicAdd(&(statsBin[currentSignature.get()]), 1 + len - KMER_LENGTH);

		return;
	}

	__global__ void copy_key(uint64_t* d_key, CKmer* d_data, int idx) {
		int tid = threadIdx.x + blockIdx.x * blockDim.x;
		d_key[tid] = d_data[tid].data[idx];
	}

	__device__ bool is_allowed(uint32_t mmer, uint32_t len)
	{
		if ((mmer & 0x3f) == 0x3f)            // TTT suffix
			return false;
		if ((mmer & 0x3f) == 0x3b)            // TGT suffix
			return false;
		if ((mmer & 0x3c) == 0x3c)            // TG* suffix
			return false;

		for (uint32_t j = 0; j < len - 3; ++j)
		if ((mmer & 0xf) == 0)                // AA inside
			return false;
		else
			mmer >>= 2;

		if (mmer == 0)            // AAA prefix
			return false;
		if (mmer == 0x04)        // ACA prefix
			return false;
		if ((mmer & 0xf) == 0)    // *AA prefix
			return false;
	
		return true;
	}

	__device__ uint32_t get_rev(uint32_t mmer, uint32_t len)
	{
		uint32_t rev = 0;
		uint32_t shift = len*2 - 2;
		for(uint32_t i = 0 ; i < len ; ++i)
		{
			rev += (3 - (mmer & 3)) << shift;
			mmer >>= 2;
			shift -= 2;
		}
		return rev;
	}



	__global__ void calc_norm(uint32_t* norm7) {
		uint32_t special = 1 << 7 * 2;
		uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	        if(i < special)
	  	{
		                uint32_t rev = get_rev(i, 7);
		                uint32_t str_val = is_allowed(i, 7) ? i : special;
		                uint32_t rev_val = is_allowed(rev, 7) ? rev : special;
		                norm7[i] = MIN(str_val, rev_val);
		}
	}

	__device__ void update_n_plus_x_recs(char* seq, uint32_t n, uint32_t &bin_desc_n_plus_x_recs) {
		unsigned char kmer, rev;
		uint32_t kmer_pos = 4;
		uint32_t rev_pos = KMER_LENGTH;
		uint32_t x;

		uint32_t n_plus_x_recs_tmp = 0;

		kmer = (seq[0] << 6) + (seq[1] << 4) + (seq[2] << 2) + seq[3];
		rev = ((3 - seq[KMER_LENGTH - 1]) << 6) + ((3 - seq[KMER_LENGTH - 2]) << 4) + ((3 - seq[KMER_LENGTH - 3]) << 2) + (3 - seq[KMER_LENGTH - 4]);

		x = 0;
		//kmer_smaller: 0, rev_smaller: 1, equals: 2
		int current_state, new_state;
		if (kmer < rev)
			current_state = 0;
		else if (rev < kmer)
			current_state = 1;
		else
			current_state = 2;


		for (uint32_t i = 0; i < n - KMER_LENGTH; ++i)
		{
			rev >>= 2;
			rev += (3 - seq[rev_pos++]) << 6;
			kmer <<= 2;
			kmer += seq[kmer_pos++];

			if (kmer < rev)
				new_state = 0;
			else if (rev < kmer)
				new_state = 1;
			else
				new_state = 2;

			if (new_state == current_state)
			{
				if (current_state == 2)
					++n_plus_x_recs_tmp;
				else
					++x;
			}
			else
			{
				current_state = new_state;
				n_plus_x_recs_tmp += 1 + x / 4;

				x = 0;
			}
		}
		n_plus_x_recs_tmp += 1 + x / 4;

		atomicAdd(&(bin_desc_n_plus_x_recs), n_plus_x_recs_tmp);
	}
	

	__device__ void putExtendedKmer(char* seq, uint32_t n, unsigned char* buffer, int64_t &bin_desc_size, uint64_t &bin_desc_n_rec
				  , uint32_t &bin_desc_n_plus_x_recs, uint64_t &bin_desc_n_super_kmers, int &lock) {
		volatile bool leaveLoop = false;
		while(!leaveLoop) 
		{
			if(atomicCAS(&lock, 0, 1) == 0)
			{
				buffer[bin_desc_size++] = n - KMER_LENGTH;		
				for(uint32_t i = 0, j = 0 ; i < n / 4 ; ++i,j+=4)
					buffer[bin_desc_size++] = (seq[j] << 6) + (seq[j + 1] << 4) + (seq[j + 2] << 2) + seq[j + 3];
				switch (n%4)
				{
					case 1:
						buffer[bin_desc_size++] = (seq[n-1] << 6);
						break;
					case 2:
						buffer[bin_desc_size++] = (seq[n-2] << 6) + (seq[n-1] << 4);
						break;
					case 3:
						buffer[bin_desc_size++] = (seq[n-3] << 6) + (seq[n-2] << 4) + (seq[n-1] << 2);
						break;
				}

				++bin_desc_n_super_kmers;
				bin_desc_n_rec += n - KMER_LENGTH + 1;						
			
				update_n_plus_x_recs(seq, n, bin_desc_n_plus_x_recs); 
				leaveLoop = true;
				atomicExch(&lock, 0);
			}		
		}
	};

	__global__ void processReads(char* readList, int* sigMap,  uint32_t* statsBin, unsigned char* buffer, int64_t* bin_desc_size, uint64_t* bin_desc_n_rec
				  , uint32_t* bin_desc_n_plus_x_recs, uint64_t* bin_desc_n_super_kmers, int* lock, int counter, uint32_t* norm7, int offset, int bin_size)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		//__shared__ shared_bins shared;

		//shared.init_bins();
		
		if(tid >= counter)
			return;
		//atomicAdd(&(statsBin[0]), 1);
		char* read = readList + (tid + offset) * NUM_BYTES_PER_READ * 4;
		
		char seq[READ_LENGTH];
		uint32_t signatureStartPos;

		CMmer currentSignature, endMmer;

		unsigned char c;
		for(int i = 0; i < READ_LENGTH; i++) {
			c = read[i];
			if(c < 32)
				break;
			seq[i] = code[c];
		}	

		uint32_t i;
		uint32_t len;//length of extended kmer

		int idx = 0;
		uint16_t lenVec[READ_LENGTH - KMER_LENGTH + 1];
		uint16_t startPosVec[READ_LENGTH - KMER_LENGTH + 1];		
		int bin_no[READ_LENGTH - KMER_LENGTH + 1];	

		i = SIGNATURE_LEN;

		len = SIGNATURE_LEN;
		signatureStartPos = i - SIGNATURE_LEN;
		currentSignature.insert(seq + signatureStartPos, norm7);
		endMmer.set(currentSignature);
		for (; i < READ_LENGTH; ++i)
		{
			endMmer.insert(seq[i], norm7);
			if (endMmer < currentSignature)//signature at the end of current k-mer is lower than current
			{
				if (len >= KMER_LENGTH)
				{
					bin_no[idx] = sigMap[currentSignature.get()];
					lenVec[idx] = len;
					startPosVec[idx++] = i - len;
					len = KMER_LENGTH - 1;
				}
				currentSignature.set(endMmer);
				signatureStartPos = i - SIGNATURE_LEN + 1;
			}
			else if (endMmer == currentSignature)
			{
				currentSignature.set(endMmer);
				signatureStartPos = i - SIGNATURE_LEN + 1;
			}
			else if (signatureStartPos + KMER_LENGTH - 1 < i)//need to find new signature
			{
				bin_no[idx] = sigMap[currentSignature.get()];
				lenVec[idx] = len;
				startPosVec[idx++] = i - len;

				len = KMER_LENGTH - 1;
				//looking for new signature
				++signatureStartPos;
				//building first signature in current k-mer
				endMmer.insert(seq + signatureStartPos, norm7);
				currentSignature.set(endMmer);
				for (uint32_t j = signatureStartPos + SIGNATURE_LEN; j <= i; ++j)
				{
					endMmer.insert(seq[j], norm7);
					if (endMmer <= currentSignature)
					{
						currentSignature.set(endMmer);
						signatureStartPos = j - SIGNATURE_LEN + 1;
					}
				}
			}
			++len;
		}
		
		if (len >= KMER_LENGTH)//last one in read
		{
			bin_no[idx] = sigMap[currentSignature.get()];	
			lenVec[idx] = len;
			startPosVec[idx++] = i - len;
		}	
	
		for(i = 0; i < idx; i++) {
			int no = bin_no[i];
			putExtendedKmer(seq + startPosVec[i], lenVec[i], buffer+bin_size * no, bin_desc_size[no]
					,bin_desc_n_rec[no],bin_desc_n_plus_x_recs[no],bin_desc_n_super_kmers[no], lock[no]);
		}
		/*
int tid = threadIdx.x + blockIdx.x * blockDim.x;

		//__shared__ shared_bins shared;

		//shared.init_bins();
		
		if(tid >= counter)
			return;
		//atomicAdd(&(statsBin[0]), 1);
		char* read = readList + tid * NUM_BYTES_PER_READ * 4;
		
		char seq[READ_LENGTH];
		uint32_t signatureStartPos;

		CMmer currentSignature, endMmer;
		int bin_no;	

		unsigned char c;
		for(int i = 0; i < READ_LENGTH; i++) {
			c = read[i];
			if(c < 32)
				break;
			seq[i] = code[c];
		}	

		uint32_t i;
		uint32_t len;//length of extended kmer

		i = 0;
		len = 0;
		while (i + KMER_LENGTH - 1 < READ_LENGTH)
		{	
			bool contains_N = false;
			//building first signature after 'N' or at the read begining
			for (uint32_t j = 0; j < SIGNATURE_LEN; ++j, ++i)
			if (seq[i] < 0)//'N'
			{
				contains_N = true;
				break;
			}
			//signature must be shorter than k-mer so if signature contains 'N', k-mer will contains it also
			if (contains_N)
			{
				++i;
				continue;
			}
			len = SIGNATURE_LEN;
			signatureStartPos = i - SIGNATURE_LEN;
			currentSignature.insert(seq + signatureStartPos, norm7);
			endMmer.set(currentSignature);
			for (; i < READ_LENGTH; ++i)
			{
				if (seq[i] < 0)//'N'
				{
					if (len >= KMER_LENGTH)
					{
						bin_no = sigMap[currentSignature.get()];	
						putExtendedKmer(seq + i - len, len, buffer+BIN_PART_SIZE * NUM_PARTS * bin_no, bin_desc_size[bin_no]
								,bin_desc_n_rec[bin_no],bin_desc_n_plus_x_recs[bin_no],bin_desc_n_super_kmers[bin_no], lock[bin_no]);
					}
					len = 0;
					++i;
					break;
				}
				endMmer.insert(seq[i], norm7);
				if (endMmer < currentSignature)//signature at the end of current k-mer is lower than current
				{
					if (len >= KMER_LENGTH)
					{
						bin_no = sigMap[currentSignature.get()];
						putExtendedKmer(seq + i - len, len, buffer+BIN_PART_SIZE * NUM_PARTS * bin_no, bin_desc_size[bin_no]
								,bin_desc_n_rec[bin_no],bin_desc_n_plus_x_recs[bin_no],bin_desc_n_super_kmers[bin_no], lock[bin_no]);

						len = KMER_LENGTH - 1;
					}
					currentSignature.set(endMmer);
					signatureStartPos = i - SIGNATURE_LEN + 1;
				}
				else if (endMmer == currentSignature)
				{
					currentSignature.set(endMmer);
					signatureStartPos = i - SIGNATURE_LEN + 1;
				}
				else if (signatureStartPos + KMER_LENGTH - 1 < i)//need to find new signature
				{
					bin_no = sigMap[currentSignature.get()];
					putExtendedKmer(seq + i - len, len, buffer+BIN_PART_SIZE * NUM_PARTS * bin_no, bin_desc_size[bin_no]
							,bin_desc_n_rec[bin_no],bin_desc_n_plus_x_recs[bin_no],bin_desc_n_super_kmers[bin_no], lock[bin_no]);

					len = KMER_LENGTH - 1;
					//looking for new signature
					++signatureStartPos;
					//building first signature in current k-mer
					endMmer.insert(seq + signatureStartPos, norm7);
					currentSignature.set(endMmer);
					for (uint32_t j = signatureStartPos + SIGNATURE_LEN; j <= i; ++j)
					{
						endMmer.insert(seq[j], norm7);
						if (endMmer <= currentSignature)
						{
							currentSignature.set(endMmer);
							signatureStartPos = j - SIGNATURE_LEN + 1;
						}
					}
				}
				++len;
				if(len == KMER_LENGTH + 255)
				{
					bin_no = sigMap[currentSignature.get()];
					putExtendedKmer(seq + i + 1 - len, len, buffer+BIN_PART_SIZE * NUM_PARTS * bin_no, bin_desc_size[bin_no]
							,bin_desc_n_rec[bin_no],bin_desc_n_plus_x_recs[bin_no],bin_desc_n_super_kmers[bin_no], lock[bin_no]);

					i -= KMER_LENGTH - 2;
					len = 0;
					break;
				}
			}
		}
		if (len >= KMER_LENGTH)//last one in read
		{
			bin_no = sigMap[currentSignature.get()];	
			putExtendedKmer(seq + i - len, len, buffer+BIN_PART_SIZE * NUM_PARTS * bin_no, bin_desc_size[bin_no]
					,bin_desc_n_rec[bin_no],bin_desc_n_plus_x_recs[bin_no],bin_desc_n_super_kmers[bin_no], lock[bin_no]);

		}	


	*/
		return;
	}

	__global__ void init_bins(unsigned char* buffer, int64_t* bin_desc_size, uint64_t* bin_desc_n_rec
				  , uint32_t* bin_desc_n_plus_x_recs, uint64_t* bin_desc_n_super_kmers, int* lock)
	{
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		if(tid > 511)
			return;

		lock[tid] = 0;
		
		bin_desc_size[tid] = 0;
		bin_desc_n_rec[tid] = 0;
		bin_desc_n_plus_x_recs[tid] = 0;
		bin_desc_n_super_kmers[tid] = 0;
		
		return;
	}


	__host__ void PreCompactKxmers(CKmer* &sort_buffer, uint64_t& compacted_count, uint32_t* &kxmer_counters, uint64_t n_plus_x_recs)
	{
		compacted_count = 0;

		CKmer *act_kmer;
		act_kmer = &sort_buffer[0];
		kxmer_counters[compacted_count] = 1;

		for (uint32_t i = 1; i < n_plus_x_recs; ++i)
		{
			if (*act_kmer == sort_buffer[i])
				++kxmer_counters[compacted_count];
			else
			{
				sort_buffer[compacted_count++] = *act_kmer;
				kxmer_counters[compacted_count] = 1;
				act_kmer = &sort_buffer[i];
			}
		}
		sort_buffer[compacted_count++] = *act_kmer;

	}

	__host__ uint64_t FindFirstSymbOccur(CKmer* sort_buffer, uint64_t start_pos, uint64_t end_pos, uint32_t offset, unsigned char symb, uint32_t max_x)
	{
		uint32_t kxmer_offset = (KMER_LENGTH + max_x - offset) * 2;
		uint64_t middle_pos;
		unsigned char middle_symb;
		while (start_pos < end_pos)
		{
			middle_pos = (start_pos + end_pos) / 2;
			middle_symb = sort_buffer[middle_pos].get_2bits(kxmer_offset);
			if (middle_symb < symb)
				start_pos = middle_pos + 1;
			else
				end_pos = middle_pos;
		}
		return end_pos;
	}

	__host__ void InitKXMerSet(CKXmerSet& kxmer_set, CKmer* sort_buffer, uint64_t start_pos, uint64_t end_pos, uint32_t offset, uint32_t depth, uint32_t max_x)
	{
		if (start_pos == end_pos)
			return;
		uint32_t shr = max_x + 1 - offset;
		kxmer_set.init_add(start_pos, end_pos, shr);

		--depth;
		if (depth > 0)
		{
			uint64_t pos[5];
			pos[0] = start_pos;
			pos[4] = end_pos;
			for (uint32_t i = 1; i < 4; ++i)
				pos[i] = FindFirstSymbOccur(sort_buffer, pos[i - 1], end_pos, offset, i, max_x);
			for (uint32_t i = 1; i < 5; ++i)
				InitKXMerSet(kxmer_set, sort_buffer, pos[i - 1], pos[i], offset + 1, depth, max_x);
		}
	}




	__host__ void compact_kxmers(CKmer* sort_buffer, uint32_t lut_prefix_len, unsigned char* buffer, uint64_t part1_size, uint64_t out_buffer_size, 
				     uint64_t in_n_plus_x_recs, uint64_t kxmers_size, uint32_t max_x, uint64_t counter_max, completerDesc &completer, int bin_no)
	{

		CKXmerSet kxmer_set(KMER_LENGTH);
		kxmer_set.clear();
		kxmer_set.set_buffer(sort_buffer);
		uint64_t n_unique = 0;
		uint64_t n_cutoff_min = 0;
		uint64_t n_cutoff_max = 0;
		uint64_t n_total = 0;

		uint32_t kmer_symbols = KMER_LENGTH - lut_prefix_len;
		uint64_t kmer_bytes = kmer_symbols / 4;
		uint64_t lut_recs = 1 << (2 * lut_prefix_len);
		uint64_t lut_size = lut_recs * sizeof(uint64_t);


		unsigned char *out_buffer = NULL;
		unsigned char *raw_lut = NULL;
		
		out_buffer = buffer + part1_size;
		raw_lut = out_buffer + out_buffer_size;

		uint64_t *lut = (uint64_t*)raw_lut;
		for(int i = 0; i < lut_recs; i++) {
			*(lut + i) = 0;
		}

		uint32_t out_pos = 0;
		uint32_t *kxmer_counters;		

		if (in_n_plus_x_recs)
		{
			unsigned char* raw_kxmer_counters = NULL;
			raw_kxmer_counters = buffer + kxmers_size;
			kxmer_counters = (uint32_t*)raw_kxmer_counters;
			uint64_t compacted_count;
			PreCompactKxmers(sort_buffer, compacted_count, kxmer_counters, in_n_plus_x_recs);

			uint64_t pos[5];//pos[symb] is first position where symb occur (at first position of k+x-mer) and pos[symb+1] jest first position where symb is not starting symbol of k+x-mer
			pos[0] = 0;
			pos[4] = compacted_count;
			for (uint32_t i = 1; i < 4; ++i)
				pos[i] = FindFirstSymbOccur(sort_buffer, pos[i - 1], compacted_count, 0, i, max_x);
			for (uint32_t i = 1; i < 5; ++i)
				InitKXMerSet(kxmer_set, sort_buffer, pos[i - 1], pos[i], max_x + 2 - i, i, max_x);

		
		

			uint64_t counter_pos = 0;
	
			uint64_t counter_size = min(BYTE_LOG(CUTOFF_MAX), BYTE_LOG(counter_max));

			CKmer kmer, next_kmer;
			CKmer kmer_mask;

			kmer.clear();
			next_kmer.clear();
	
			kmer_mask.set_n_1(KMER_LENGTH * 2);
			uint32_t count;
			//first
			kxmer_set.get_min(counter_pos, kmer);
			count = kxmer_counters[counter_pos];
			//rest
			while (kxmer_set.get_min(counter_pos, next_kmer))
			{
				if (kmer == next_kmer)
					count += kxmer_counters[counter_pos];
				else
				{
					n_total += count;
					++n_unique;
					if (count < (uint32_t)CUTOFF_MIN)
						n_cutoff_min++;
					else if (count >(uint32_t)CUTOFF_MAX)
						n_cutoff_max++;
					else
					{
						lut[kmer.remove_suffix(2 * kmer_symbols)]++;
						if (count > (uint32_t)counter_max)
							count = counter_max;
	
						// Store compacted kmer

						for (int32_t j = (int32_t)kmer_bytes - 1; j >= 0; --j)
							out_buffer[out_pos++] = kmer.get_byte(j);
						for (int32_t j = 0; j < (int32_t)counter_size; ++j)
							out_buffer[out_pos++] = (count >> (j * 8)) & 0xFF;
					}
					count = kxmer_counters[counter_pos];
					kmer = next_kmer;
				}
			}

			//last one
			++n_unique;
			n_total += count;
			if (count < (uint32_t)CUTOFF_MIN)
				n_cutoff_min++;
			else if (count >(uint32_t)CUTOFF_MAX)
				n_cutoff_max++;
			else
			{
				lut[kmer.remove_suffix(2 * kmer_symbols)]++;
				if (count > (uint32_t)counter_max)
					count = counter_max;
	
				// Store compacted kmer
				for (int32_t j = (int32_t)kmer_bytes - 1; j >= 0; --j)
					out_buffer[out_pos++] = kmer.get_byte(j);
				for (int32_t j = 0; j < (int32_t)counter_size; ++j)
					out_buffer[out_pos++] = (count >> (j * 8)) & 0xFF;
			}

		}
		completer.data[bin_no] = out_buffer;
		completer.data_size[bin_no] = out_pos;
		completer.lut[bin_no] = raw_lut;
		completer.lut_size[bin_no] = lut_size;
		completer._n_unique[bin_no] = n_unique;
		completer._n_cutoff_min[bin_no] = n_cutoff_min;
		completer._n_cutoff_max[bin_no] = n_cutoff_max;
		completer._n_total[bin_no] = n_total;
	}

	__host__ bool store_uint(FILE *out, uint64_t x, uint32_t size)
	{
		for(uint32_t i = 0; i < size; ++i)
			putc((x >> (i * 8)) & 0xFF, out);

		return true;
	}


	int main(int argc, char** argv) {
		//cuda variables
		std::ios_base::sync_with_stdio(false);
		hipError_t cerr;
		const char* cudaErrorStr;
		
		//fastq file
		FILE* fastqFile;
		
		//output file
		int reads_per_load = READS_PER_LOAD /** KMER_LENGTH / 44*/;
		/*if(KMER_LENGTH > 44)
			reads_per_load = READS_PER_LOAD;
		*/
		
		//host variables
		char* read                     = (char*) malloc(sizeof(char) * (READ_LENGTH + 2));
		char* dumm                     = (char*) malloc(sizeof(char) * (READ_LENGTH * 2 + 1));
		char* statsReadListHost        = (char*) malloc(sizeof(char) * STATS_READ_NUM * NUM_BYTES_PER_READ * 4);
		char* readListHost	       = (char*) malloc(sizeof(char) * reads_per_load * NUM_BYTES_PER_READ * 4);
		uint32_t* binStatsHost	       = (uint32_t*) malloc(sizeof(uint32_t) * ((1 << SIGNATURE_LEN * 2) + 1));
		int signature_map[MAP_SIZE];
		char* codesHost		       = (char*) malloc(sizeof(char) * CODES_SIZE);	
		binDesc binDescHost;
		FILE* binFiles[BIN_NO];
		
		unsigned char* buffer_host;//	= (unsigned char*)malloc(sizeof(unsigned char)*(BIN_PART_SIZE * NUM_PARTS * BIN_NO));
		int64_t* bin_desc_size_host;//	= (int64_t*)malloc(sizeof(int64_t)*BIN_NO*NUM_STREAM);
		uint64_t* bin_desc_n_rec_host;//	= (uint64_t*)malloc(sizeof(uint64_t)*BIN_NO*NUM_STREAM);
		uint32_t* bin_desc_n_plus_x_recs_host;// = (uint32_t*)malloc(sizeof(uint32_t)*BIN_NO*NUM_STREAM);
		uint64_t* bin_desc_n_super_kmers_host;// = (uint64_t*)malloc(sizeof(uint64_t)*BIN_NO*NUM_STREAM);

		//stage two variables

		//device variables
		char* statsReadDevice;	
		char* readListDevice;
		uint32_t* binStatsDevice;
		int* sigMapDevice;
		uint32_t* norm7;
		unsigned char* buffer_device;
		int64_t* bin_desc_size_device;
		uint64_t* bin_desc_n_rec_device;
		uint32_t* bin_desc_n_plus_x_recs_device;
		uint64_t* bin_desc_n_super_kmers_device;
		int* lock_device;

		int fileAlive;
		int readCounter;

		hipHostAlloc((void**)&buffer_host, (unsigned long)(sizeof(unsigned char)*(BIN_PART_SIZE * NUM_PARTS * BIN_NO)), 0);
		hipHostAlloc((void**)&bin_desc_size_host, (unsigned long)(sizeof(int64_t)*(NUM_STREAM * BIN_NO)), 0);
		hipHostAlloc((void**)&bin_desc_n_rec_host, (unsigned long)(sizeof(uint64_t)*(NUM_STREAM * BIN_NO)), 0);
		hipHostAlloc((void**)&bin_desc_n_plus_x_recs_host, (unsigned long)(sizeof(uint32_t)*(NUM_STREAM * BIN_NO)), 0);
		hipHostAlloc((void**)&bin_desc_n_super_kmers_host, (unsigned long)(sizeof(uint64_t)*(NUM_STREAM * BIN_NO)), 0);


		double processReadsTimer = 0;	

		timeval timeStart, timeEnd;
		gettimeofday(&(timeStart), NULL);
		deletefiles();
		cout<<"kmer length is:"<<KMER_LENGTH<<endl;
		for(int i = 0; i < MAP_SIZE; i++)
			signature_map[i] = -1;

		size_t mem_tot, mem_free;
		
		fileAlive = 1;
		readCounter = 0;
		fastqFile = fopen(argv[1], "r");
		//outFile = fopen(argv[2], "w");
		//read fastq reads into stats read buffer
		while(fileAlive > 0)
		{
			if (fileAlive > 0){ 
				if(fgets(dumm, READ_LENGTH * 2 + 1, fastqFile) == NULL)
					fileAlive = 0;
			}
			if (fileAlive > 0){ 
				if(fgets(read, READ_LENGTH + 2, fastqFile) == NULL)
					fileAlive = 0;
			}
			if (fileAlive > 0){ 
				if(fgets(dumm, READ_LENGTH * 2 + 1, fastqFile) == NULL)
					fileAlive = 0;
			}
			if (fileAlive > 0){ 
				if(fgets(dumm, READ_LENGTH * 2 + 1, fastqFile) == NULL)
					fileAlive = 0;
			}
			if (fileAlive > 0)
			{
				//fprintf(outFile, "%s", read);
				memcpy(statsReadListHost + NUM_BYTES_PER_READ * 4 * readCounter, read, READ_LENGTH);
				readCounter++;

				if(readCounter >= STATS_READ_NUM)
					break;
			}
		}
		fclose(fastqFile);
		//set device to device 0	
		cerr = hipSetDevice(0);
		if(cerr != hipSuccess) {
			printf("cannot set to device 0, %s\n", cerr);
			return 1;
		}
		//prepare codes for constant memory
		calc_codes(codesHost);	

		cerr = hipMemcpyToSymbol(HIP_SYMBOL(code), codesHost, CODES_SIZE * sizeof(char));
		if(cerr != hipSuccess) {
			printf("cannot copy codes to symbol\n");
			return 1;
		}
		//allocate space for stats fastq reads
		cerr = hipMalloc(&statsReadDevice, sizeof(char) * STATS_READ_NUM * NUM_BYTES_PER_READ * 4);
		if(cerr != hipSuccess) {
			printf("cannot allocate memory for stats reads on device\n");
			return 1;
		}

		cerr = hipMalloc((void**)&readListDevice, sizeof(char) * reads_per_load * NUM_BYTES_PER_READ * 4);
		if(cerr != hipSuccess) {
			printf("cannot allocate memory for reads list on device\n");
			return 1;
		}

		//allocate space for signature map
		cerr = hipMalloc(&sigMapDevice, sizeof(int) * MAP_SIZE);
		if(cerr != hipSuccess) {
			printf("cannot allocate memory for signature map on device\n");
			return 1;
		}

		cerr = hipMalloc(&norm7, sizeof(int) * (1<<7*2));
		if(cerr != hipSuccess) {
			printf("cannot allocate memory for norm7  on device\n");
			return 1;
		}

		
		//allocate space for stats
		cerr = hipMalloc(&binStatsDevice, sizeof(uint32_t) * ((1 << SIGNATURE_LEN * 2) + 1));
		if(cerr != hipSuccess) {
			printf("cannot allocate memory for stats bins on device\n");
			return 1;
		}
	
		cerr = hipMalloc(&buffer_device, sizeof(char) * (BIN_PART_SIZE * NUM_PARTS * BIN_NO));
		if(cerr != hipSuccess) {
			printf("cannot allocate memory for buffer_device on device\n");
			return 1;
		}

		cerr = hipMalloc(&bin_desc_size_device, sizeof(int64_t) * (BIN_NO*NUM_STREAM));
		if(cerr != hipSuccess) {
			printf("cannot allocate memory for bin_desc_size_device on device\n");
			return 1;
		}

		cerr = hipMalloc(&bin_desc_n_rec_device, sizeof(uint64_t) * (BIN_NO*NUM_STREAM));
		if(cerr != hipSuccess) {
			printf("cannot allocate memory for bin_desc_n_rec on device\n");
			return 1;
		}

		cerr = hipMalloc(&bin_desc_n_plus_x_recs_device, sizeof(uint32_t) * (BIN_NO*NUM_STREAM));
		if(cerr != hipSuccess) {
			printf("cannot allocate memory for bin_desc_n_plus_x_recs on device\n");
			return 1;
		}

		cerr = hipMalloc(&bin_desc_n_super_kmers_device, sizeof(uint64_t) * (BIN_NO*NUM_STREAM));
		if(cerr != hipSuccess) {
			printf("cannot allocate memory for bin_desc_n_super_kmers on device\n");
			return 1;
		}

		cerr = hipMalloc(&lock_device, sizeof(int) * (BIN_NO*NUM_STREAM));
		if(cerr != hipSuccess) {
			printf("cannot allocate memory for lock on device\n");
			return 1;
		}


		//initialize stats bins to 0
		cerr = hipMemset(binStatsDevice, 0, sizeof(uint32_t) * ((1 << SIGNATURE_LEN * 2) + 1));
		if(cerr != hipSuccess) {
			printf("cannot initialize statsBins to 0\n");
			return 1;
		}
		
		
		//transfer stats fastq reads to device
		cerr = hipMemcpy(statsReadDevice, statsReadListHost, sizeof(char) * STATS_READ_NUM * NUM_BYTES_PER_READ * 4, hipMemcpyHostToDevice);
		if(cerr != hipSuccess) {
			cudaErrorStr = hipGetErrorString(cerr);
			printf("cannot copy stats read list to device: %s\n", cudaErrorStr); 
			return 1;
		}

		
		calc_norm<<<(1<<7*2)/NUM_THREADS_PER_BLOCK, NUM_THREADS_PER_BLOCK>>>(norm7);


		
		//execution of the calcStats kernel
		calcStats<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(statsReadDevice, binStatsDevice, norm7);	
		printf("calcStats done\n");
		cerr = hipMemcpy(binStatsHost, binStatsDevice, sizeof(uint32_t) * MAP_SIZE, hipMemcpyDeviceToHost);
		if(cerr != hipSuccess) {
			cudaErrorStr = hipGetErrorString(cerr);
			printf("cannot copy binstats back to host: %s\n", cudaErrorStr); 
			return 1;
		}
		
		init(binStatsHost,signature_map);


		cerr = hipMemcpy(sigMapDevice, signature_map, sizeof(int) * MAP_SIZE, hipMemcpyHostToDevice);
		if(cerr != hipSuccess) {
			cudaErrorStr = hipGetErrorString(cerr);
			printf("cannot copy signature map to device: %s\n", cudaErrorStr); 
			return 1;
		}

		cerr = hipMemcpy(signature_map,sigMapDevice, sizeof(int) * MAP_SIZE, hipMemcpyDeviceToHost);

		//printf("16384 is: %d", signature_map[16384]);
		//preprocessing stage finished
		hipFree(statsReadDevice);	
		//stage 1

		uint64_t n_reads = 0;

		uint64_t tmp_size = 0;

		fastqFile = fopen(argv[1], "r");

		double fastqReaderTimer = 0;

		int num_block = ceil((double)reads_per_load /(double)NUM_THREADS_PER_BLOCK);
		printf("num_block is: %d\n", num_block);
		
		printf("reads_per_load is: %d\n", reads_per_load);


		readCounter = 0;
		


		init_bin_desc(binDescHost);
		fileAlive = 1;

		int ctr = 0;

		while(fileAlive > 0)
		{
			if (fileAlive > 0){ 
				if(fgets(dumm, READ_LENGTH * 2 + 1, fastqFile) == NULL)
					fileAlive = 0;
			}
			if (fileAlive > 0){ 	
				if(fgets(read, READ_LENGTH + 2, fastqFile) == NULL)
					fileAlive = 0;
			}
			if (fileAlive > 0){ 
				if(fgets(dumm, READ_LENGTH * 2 + 1, fastqFile) == NULL)
					fileAlive = 0;
			}
			if (fileAlive > 0){ 
				if(fgets(dumm, READ_LENGTH * 2 + 1, fastqFile) == NULL)
					fileAlive = 0;
			}
			if (fileAlive > 0)
			{
				memcpy(readListHost + NUM_BYTES_PER_READ * 4 * readCounter, read, READ_LENGTH);
				readCounter++;
				ctr++;
				if(readCounter >= reads_per_load)
					break;
			}
		}

		n_reads += readCounter;
		int next_batch = fileAlive;
		int batch_num = 0;

		hipStream_t stream[NUM_STREAM];
	
		for(int i = 0; i < NUM_STREAM; i++) 
			hipStreamCreate(&stream[i]);
		
		int buffer_size_per_stream = BIN_PART_SIZE * NUM_PARTS * BIN_NO / NUM_STREAM; 
		int bin_size = buffer_size_per_stream / BIN_NO;
		printf("buffer size is %d\n", buffer_size_per_stream);
		while(next_batch > 0) {
			if(fileAlive <= 0)
				next_batch = 0;
			
			int readNum = readCounter / NUM_STREAM;
			int offset;
			offset = 0;
			if(0 == NUM_STREAM - 1)
				readNum = reads_per_load;
		
			cerr = hipMemcpyAsync(readListDevice + offset, readListHost + offset, sizeof(char) * readNum * NUM_BYTES_PER_READ * 4, hipMemcpyHostToDevice, stream[0]);
			if(cerr != hipSuccess) {
				cudaErrorStr = hipGetErrorString(cerr);
				printf("cannot copy read list to device: %s\n", cudaErrorStr); 
				printf("%d", readCounter);
				return 1;
			}
			

			num_block = ceil((double)(readCounter / NUM_STREAM) / (double)NUM_THREADS_PER_BLOCK);
			
			init_bins<<<1, BIN_NO, 0, stream[0]>>>(buffer_device, bin_desc_size_device, bin_desc_n_rec_device
								, bin_desc_n_plus_x_recs_device, bin_desc_n_super_kmers_device,lock_device);
			
			readNum = readCounter / NUM_STREAM ;
			
			processReads<<<num_block, NUM_THREADS_PER_BLOCK, 0, stream[0]>>>(readListDevice, sigMapDevice, binStatsDevice, buffer_device
						, bin_desc_size_device, bin_desc_n_rec_device, bin_desc_n_plus_x_recs_device
						, bin_desc_n_super_kmers_device, lock_device, readNum, norm7, 0, bin_size);	
			
			if(batch_num != 0)
				putBinsToDisk(binFiles, tmp_size, binDescHost, buffer_host, bin_desc_size_host, bin_desc_n_rec_host
						, bin_desc_n_plus_x_recs_host, bin_desc_n_super_kmers_host, buffer_size_per_stream, bin_size);

			int reads_per_stream = reads_per_load / (NUM_STREAM - 1);
			int newReadCounter = 0;	
	
			for(int i = 1; i < NUM_STREAM + 1; i++) {
				int new_offset, new_readNum;
				if(i != NUM_STREAM) {	
					new_offset = readNum * NUM_BYTES_PER_READ * 4 * i;
					if(i == NUM_STREAM - 1)
						readNum = reads_per_load - readNum * (NUM_STREAM - 1);
				
					cerr = hipMemcpyAsync(readListDevice + new_offset, readListHost + new_offset, sizeof(char) * readNum * NUM_BYTES_PER_READ * 4, hipMemcpyHostToDevice, stream[i]);
					if(cerr != hipSuccess) {
						cudaErrorStr = hipGetErrorString(cerr);
						printf("cannot copy read list to device: %s\n", cudaErrorStr); 
						printf("%d", readCounter);
						return 1;
					}
					
					new_offset = BIN_NO * i;
					init_bins<<<1, BIN_NO, 0, stream[i]>>>(buffer_device, bin_desc_size_device + new_offset, bin_desc_n_rec_device + new_offset
										, bin_desc_n_plus_x_recs_device + new_offset, bin_desc_n_super_kmers_device + new_offset,lock_device + new_offset);
					
					new_readNum = readCounter / NUM_STREAM ;
					if(i == NUM_STREAM - 1)
						new_readNum = readCounter - new_readNum * (NUM_STREAM - 1);
					
					processReads<<<num_block, NUM_THREADS_PER_BLOCK, 0, stream[i]>>>(readListDevice, sigMapDevice, binStatsDevice, buffer_device + i * buffer_size_per_stream
								, bin_desc_size_device + new_offset, bin_desc_n_rec_device + new_offset, bin_desc_n_plus_x_recs_device + new_offset
								, bin_desc_n_super_kmers_device + new_offset, lock_device + new_offset, new_readNum, norm7, (readCounter/NUM_STREAM) * i, bin_size);	
				}
				//PUSH_RANGE("buffer", 3)	
				cerr = hipMemcpyAsync(buffer_host + (i - 1) * buffer_size_per_stream, buffer_device + (i - 1) * buffer_size_per_stream, sizeof(char) * (buffer_size_per_stream), hipMemcpyDeviceToHost, stream[i - 1]);
				if(cerr != hipSuccess) {
					cudaErrorStr = hipGetErrorString(cerr);
					printf("cannot copy buffer to host: %s\n", cudaErrorStr); 
					printf("%d", readCounter);
					return 1;
				}
				//POP_RANGE
				if(i != NUM_STREAM) {
					int reads_limit = (i == NUM_STREAM - 1)?reads_per_load:(reads_per_stream * i);	
					PUSH_RANGE("fget", 2)
					while(fileAlive > 0)
					{
						if (fileAlive > 0){ 
							if(fgets(dumm, READ_LENGTH * 2 + 1, fastqFile) == NULL)
								fileAlive = 0;
						}
						if (fileAlive > 0){ 
							if(fgets(read, READ_LENGTH + 2, fastqFile) == NULL)
								fileAlive = 0;
						}
						if (fileAlive > 0){ 
							if(fgets(dumm, READ_LENGTH * 2 + 1, fastqFile) == NULL)
								fileAlive = 0;
						}
						if (fileAlive > 0){ 
							if(fgets(dumm, READ_LENGTH * 2 + 1, fastqFile) == NULL)
								fileAlive = 0;
						}
						if (fileAlive > 0)
						{
							memcpy(readListHost + NUM_BYTES_PER_READ * 4 * newReadCounter, read, READ_LENGTH);
							newReadCounter++;
							ctr++;
							if(newReadCounter >= reads_limit)
								break;
						}
					}
					POP_RANGE
				}
				cerr = hipMemcpyAsync(bin_desc_size_host + offset, bin_desc_size_device + offset, sizeof(int64_t) * (BIN_NO), hipMemcpyDeviceToHost, stream[i - 1]);
				if(cerr != hipSuccess) {
					cudaErrorStr = hipGetErrorString(cerr);
					printf("cannot copy bin_desc_size to host: %s %d\n", cudaErrorStr, offset); 
					return 1;
				}
		
				cerr = hipMemcpyAsync(bin_desc_n_rec_host + offset, bin_desc_n_rec_device + offset, sizeof(uint64_t) * (BIN_NO), hipMemcpyDeviceToHost, stream[i - 1]);
				if(cerr != hipSuccess) {
					cudaErrorStr = hipGetErrorString(cerr);
					printf("cannot copy bin_desc_n_rec to host: %s\n", cudaErrorStr); 
					return 1;
				}
		
				cerr = hipMemcpyAsync(bin_desc_n_plus_x_recs_host + offset, bin_desc_n_plus_x_recs_device + offset, sizeof(uint32_t) * (BIN_NO), hipMemcpyDeviceToHost, stream[i - 1]);
				if(cerr != hipSuccess) {
					cudaErrorStr = hipGetErrorString(cerr);
					printf("cannot copy bin_desc_n_plus_x_recs to host: %s\n", cudaErrorStr); 
					return 1;
				}
		
				cerr = hipMemcpyAsync(bin_desc_n_super_kmers_host + offset, bin_desc_n_super_kmers_device + offset, sizeof(uint64_t) * (BIN_NO), hipMemcpyDeviceToHost, stream[i - 1]);
				if(cerr != hipSuccess) {
					cudaErrorStr = hipGetErrorString(cerr);
					printf("cannot copy bin_desc_n_super_kmers to host: %s\n", cudaErrorStr); 
					return 1;
				}
				offset = new_offset;
				
			}
			
			hipDeviceSynchronize();
 			readCounter = newReadCounter;

			batch_num++;
			printf("batch done\n");
		}

		putBinsToDisk(binFiles, tmp_size, binDescHost, buffer_host, bin_desc_size_host, bin_desc_n_rec_host, bin_desc_n_plus_x_recs_host, bin_desc_n_super_kmers_host
				, buffer_size_per_stream, bin_size);
		
		gettimeofday(&(timeEnd), NULL);
		timeval res;
		timersub(&(timeEnd), &(timeStart), &res);
		processReadsTimer += res.tv_sec + res.tv_usec/1000000.0; 	
		
		for(int i = 0; i < NUM_STREAM; i++) {
			hipStreamDestroy(stream[i]);
		}
		
		//free memory after 1st stage
		hipFree(readListDevice);
		hipFree(binStatsDevice);
		hipFree(sigMapDevice);
		hipFree(buffer_device);
		hipFree(bin_desc_size_device);
		hipFree(bin_desc_n_rec_device);
		hipFree(bin_desc_n_plus_x_recs_device);
		hipFree(bin_desc_n_super_kmers_device);
		hipFree(lock_device);
		hipFree(norm7);
		hipHostFree(buffer_host);
		hipHostFree(bin_desc_size_host);
		hipHostFree(bin_desc_n_rec_host);
		hipHostFree(bin_desc_n_plus_x_recs_host);
		hipHostFree(bin_desc_n_super_kmers_host);
		
		printf("fastq reader timer is:%f \n", fastqReaderTimer); 
		printf("process read timer is:%f \n", processReadsTimer);
		cout<<"tmp size is: "<<tmp_size<<endl;
		/*
		for(int i = 0; i < BIN_NO; i++) {
			fprintf(outFile, "%d %d %d %d\n", binDescHost.bin_desc_size[i], binDescHost.bin_desc_n_rec[i], binDescHost.bin_desc_n_plus_x_recs[i],binDescHost.bin_desc_n_super_kmers[i]);
		}		
		*/
		
		//start stage 2
		printf("stage 2 started\n");

		double second_timer = 0;
		gettimeofday(&(timeStart), NULL);


		//host variables
		uint32_t max_x;
		uint64_t counter_max = 255;

		//device variables	
		//uint64_t *startArrayDevice;
		//uint64_t *posArrayDevice;


		rewindfiles(binFiles);

		/*cerr = hipMalloc(&startArrayDevice, sizeof(uint64_t) * POS_ARRAY_SIZE);
		if(cerr != hipSuccess) {
			printf("cannot allocate memory for start array on device\n");
			return 1;
		}

		cerr = hipMalloc(&posArrayDevice, sizeof(uint64_t) * POS_ARRAY_SIZE);
		if(cerr != hipSuccess) {
			printf("cannot allocate memory for pos array on device\n");
			return 1;
		}*/

		//building lookuptable
		unsigned char lut[256];
		for(int i = 0; i < 256; i++)
			lut[i] = ((3 - (i & 3)) << 6) + ((3 - ((i >> 2) & 3)) << 4) + ((3 - ((i >> 4) & 3)) << 2) + (3 - ((i >> 6) & 3));



		//calculate lut prefix len
		uint32_t best_lut_prefix_len = 0;
		uint64_t best_mem_amount = 1ull << 62;

		uint32_t lut_prefix_len;

		for (lut_prefix_len = 2; lut_prefix_len < 16; ++lut_prefix_len)
		{
			uint32_t suffix_len = KMER_LENGTH - lut_prefix_len;
			if (suffix_len % 4)
				continue;

			uint64_t est_suf_mem = n_reads * suffix_len;
			uint64_t lut_mem = BIN_NO * (1ull << (2 * lut_prefix_len)) * sizeof(uint64_t);

			if (est_suf_mem + lut_mem < best_mem_amount)
			{
				best_lut_prefix_len = lut_prefix_len;
				best_mem_amount = est_suf_mem + lut_mem;
			}
		}

		lut_prefix_len = best_lut_prefix_len;

		if(KMER_LENGTH % 32 == 0)
			max_x = 0;
		else
			max_x = MIN(31 -(KMER_LENGTH % 32), KMER_X);


		printf("kmerword is: %d\n", (int)KMER_WORDS);
		printf("n_reads is: %ld\n", n_reads);
		printf("lut prefix len is: %d\n", lut_prefix_len);

		//open file for storing result
		string kmer_file_name = "result.kmc_suf";
		string lut_file_name = "result.kmc_pre";


		FILE *out_kmer = fopen(kmer_file_name.c_str(), "wb");
		if(!out_kmer)
		{
			cout << "Error: Cannot create " << kmer_file_name << "\n";
			exit(1);
		
		}

		FILE *out_lut = fopen(lut_file_name.c_str(), "wb");
		if(!out_lut)
		{
			cout << "Error: Cannot create " << lut_file_name << "\n";
			fclose(out_kmer);
			exit(1);
		}

		char s_kmc_pre[] = "KMCP";
		char s_kmc_suf[] = "KMCS";

		// Markers at the beginning
		fwrite(s_kmc_pre, 1, 4, out_lut);
		fwrite(s_kmc_suf, 1, 4, out_kmer);


		completerDesc completer;

		uint64_t n_unique, n_cutoff_min, n_cutoff_max, n_total;
		n_unique  = n_cutoff_min  = n_cutoff_max  = n_total  = 0;

		uint32_t *sig_map = new uint32_t[MAP_SIZE];
		for(int i = 0; i < MAP_SIZE; i++)
			sig_map[i] = 0;
		uint32_t lut_pos = 0;
		uint64_t n_recs = 0;

		uint64_t data_size = 0;
		int counter = 0;
		hipMemGetInfo(&mem_free, &mem_tot);
		std::cout << "Free memory : " << mem_free <<"total memory : "<<mem_tot << std::endl;

		
		int input_pos_sum = 0;
	

		omp_lock_t lock;
		omp_init_lock(&lock);	
		#pragma omp parallel for num_threads(1)	
		for(int i = 0; i < BIN_NO; i++) {
			//calculate sizes
			uint64_t startArrayHost[POS_ARRAY_SIZE];
			uint64_t posArrayHost[POS_ARRAY_SIZE];
			int arrayIdx = 0;

			
			if(binDescHost.bin_desc_size[i] == 0)	
				continue;
			uint64_t input_kmer_size;
			uint64_t kxmer_counter_size;

			input_kmer_size = binDescHost.bin_desc_n_plus_x_recs[i] * sizeof(CKmer);
			kxmer_counter_size = binDescHost.bin_desc_n_plus_x_recs[i] * sizeof(uint32_t);	

			uint64_t max_out_recs    = (binDescHost.bin_desc_n_rec[i]+1) / max(CUTOFF_MIN, 1);	
			
			uint64_t counter_size    = min(BYTE_LOG(CUTOFF_MAX), BYTE_LOG(counter_max));

			uint32_t kmer_symbols = KMER_LENGTH - lut_prefix_len;
			uint64_t kmer_bytes = kmer_symbols / 4;
			uint64_t out_buffer_size = max_out_recs * (kmer_bytes + counter_size);
				

			uint64_t lut_recs = 1 << (2 * lut_prefix_len);
			uint64_t lut_size = lut_recs * sizeof(uint64_t);
			
			uint64_t part1_size;
			uint64_t part2_size;

			part1_size = max(round_up_to_alignment(input_kmer_size) + round_up_to_alignment(kxmer_counter_size), round_up_to_alignment(binDescHost.bin_desc_size[i]));
			part2_size = max(round_up_to_alignment(input_kmer_size), round_up_to_alignment(out_buffer_size) + round_up_to_alignment(lut_size));
			uint64_t req_size = part1_size + part2_size;

			unsigned char* buffer = (unsigned char*)malloc((round_up_to_alignment(req_size) + ALIGNMENT) * sizeof(unsigned char));

			unsigned char* file_buffer;
			file_buffer = buffer;
			//read files
			fread(file_buffer, 1, binDescHost.bin_desc_size[i], binFiles[i]);	
			//calculate pos arrays
			uint32_t num_threads = 1;
			uint32_t thread_no = 0;
			uint64_t bytes_per_thread = (binDescHost.bin_desc_size[i] + num_threads - 1) / num_threads;	

			uint64_t start = 0;
			uint64_t pos = 0;

			uint32_t input_pos = 0;
			
			for(; pos < binDescHost.bin_desc_size[i]; pos += 1 + (file_buffer[pos] + KMER_LENGTH + 3) / 4) {
				if ((thread_no + 1) * bytes_per_thread <= pos)
				{
					startArrayHost[arrayIdx] = start;
					posArrayHost[arrayIdx] = pos;
					arrayIdx++;
					start = pos;
					++thread_no;
				}
			}
			if (start < pos)
			{	startArrayHost[arrayIdx] = start;
				posArrayHost[arrayIdx] = binDescHost.bin_desc_size[i];
				arrayIdx++;
			}
			CKmer * kmer_arrays;
			
			kmer_arrays = reinterpret_cast<CKmer *>(buffer + part1_size);
			thrust::host_vector<CKmer> H_data;  			


			for(int index = 0; index < arrayIdx; index++) {
				expandKxmers(file_buffer, kmer_arrays, startArrayHost[index], posArrayHost[index], max_x, lut, input_pos);
			}
			
			if(binDescHost.bin_desc_size[i] != 0) {
				input_pos_sum += input_pos;
				counter += posArrayHost[arrayIdx -1];
			}
			arrayIdx = 0;
	
			CKmer * result_arrays;
			result_arrays = reinterpret_cast<CKmer *>(buffer);				

			uint64_t sort_rec = input_pos;
	
			omp_set_lock(&lock);
			CKmer* d_data;
			cerr = hipMalloc((void**)&d_data, sizeof(CKmer) * sort_rec);
			if(cerr != hipSuccess) {
				printf("cannot allocate memory for d_data on device\n");
				exit(1);
			}

			//printf("size of d_data is: %d\n", sizeof(CKmer) * sort_rec);
			cerr = hipMemcpy(d_data, kmer_arrays, sizeof(CKmer) * sort_rec, hipMemcpyHostToDevice);
			if(cerr != hipSuccess) {
				cudaErrorStr = hipGetErrorString(cerr);
				printf("cannot d_data to device: %s\n", cudaErrorStr); 
				exit(1);
			}
		
			uint64_t *d_key;
			cerr = hipMalloc((void**)&d_key, sizeof(uint64_t) * sort_rec);
			if(cerr != hipSuccess) {
				printf("cannot allocate memory for d_key on device\n");
				exit(1);
			}

			num_block = ceil((double)sort_rec /(double)1024);
			
			//sort the kxmers			

			for(int sort_idx = 0; sort_idx < KMER_WORDS; sort_idx++) {
				copy_key<<<num_block, 1024>>>(d_key, d_data, sort_idx);
				thrust::device_ptr<CKmer> data_ptr(d_data);
				thrust::device_ptr<uint64_t> key_ptr(d_key);
				
				thrust::sort_by_key(key_ptr, key_ptr + sort_rec, data_ptr);
				hipDeviceSynchronize();
			}
			cerr = hipDeviceSynchronize();
			if(cerr != hipSuccess) {
				cudaErrorStr = hipGetErrorString(cerr);
				printf("cuda error is: %s\n", cudaErrorStr); 
				exit(1);
			}

			cerr = hipMemcpy(result_arrays, d_data, sizeof(CKmer) * sort_rec, hipMemcpyDeviceToHost);
			if(cerr != hipSuccess) {
				cudaErrorStr = hipGetErrorString(cerr);
				printf("cannot d_data back to host: %s\n", cudaErrorStr); 
				exit(1);
			}
			hipFree(d_data);
			hipFree(d_key);			

			omp_unset_lock(&lock);
			
			compact_kxmers(result_arrays, lut_prefix_len, buffer, part1_size, out_buffer_size, 
				       sort_rec, input_kmer_size, max_x, counter_max, completer, i);
			
			//compact the kxmers
			data_size += completer.data_size[i];				
		
			lut_recs = completer.lut_size[i] / sizeof(uint64_t);

			omp_set_lock(&lock);
			fwrite(completer.data[i], 1, completer.data_size[i], out_kmer);

			uint64_t *ulut = (uint64_t*)completer.lut[i];
			for(uint64_t j = 0; j < lut_recs; ++j)
			{
				uint64_t x = ulut[j];
				ulut[j] = n_recs;
				n_recs += x;
			}
			fwrite(completer.lut[i], lut_recs, sizeof(uint64_t), out_lut);		
	
			omp_unset_lock(&lock);

			n_unique     += completer._n_unique[i];
			n_cutoff_min += completer._n_cutoff_min[i];
			n_cutoff_max += completer._n_cutoff_max[i];
			n_total      += completer._n_total[i];
			for (uint32_t j = 0; j < MAP_SIZE; ++j)
			{
				if (signature_map[j] == i)
				{
					sig_map[j] = lut_pos;
				}
			}
			++lut_pos;
			free(buffer); 
		}

		// Marker at the end
		fwrite(s_kmc_suf, 1, 4, out_kmer);
		fclose(out_kmer);

		fwrite(&n_recs, 1, sizeof(uint64_t), out_lut);

		//store signature mapping 
		fwrite(sig_map, sizeof(uint32_t), MAP_SIZE, out_lut);	

		uint64_t counter_size = counter_size = min(BYTE_LOG(CUTOFF_MAX), BYTE_LOG(counter_max));
		// Store header
		uint32_t offset = 0;

		store_uint(out_lut, KMER_LENGTH, 4);				offset += 4;
		store_uint(out_lut, (uint32_t)false, 4);			offset += 4;	// mode: 0 (counting), 1 (Quake-compatibile counting)
		store_uint(out_lut, counter_size, 4);				offset += 4;
		store_uint(out_lut, lut_prefix_len, 4);				offset += 4;
		store_uint(out_lut, SIGNATURE_LEN, 4);				offset += 4; 
		store_uint(out_lut, CUTOFF_MIN, 4);				offset += 4;
		store_uint(out_lut, CUTOFF_MAX, 4);				offset += 4;
		store_uint(out_lut, n_unique - n_cutoff_min - n_cutoff_max, 8);		offset += 8;

		// Space for future use
		for(int32_t i = 0; i < 7; ++i)
		{
			store_uint(out_lut, 0, 4);
			offset += 4;
		}
	
		store_uint(out_lut, 0x200, 4);
		offset += 4;

		store_uint(out_lut, offset, 4);

		// Marker at the end
		fwrite(s_kmc_pre, 1, 4, out_lut);
		fclose(out_lut);
			
		gettimeofday(&(timeEnd), NULL);
		timersub(&(timeEnd), &(timeStart), &res);
		second_timer += res.tv_sec + res.tv_usec/1000000.0; 	

		cout<<"second stage timer is: "<<second_timer<<endl;

		printf("input_pos sum is: %d\n", input_pos_sum);

		closefiles(binFiles);
		deletefiles();

		//fclose(outFile);
		printf("done\n");
		return 0;

}
